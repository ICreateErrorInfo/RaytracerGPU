#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <bitset>
#include <fstream>

#include <iostream>
#include <fstream>
#include "vec3.h"
#include "ray.h"
#include "hitable.h"
#include <hiprand/hiprand_kernel.h>
#include "hitable_list.h"
#include "sphere.h"
#include "camera.h"
#include "material.h"
#include "aarect.h"
#include "moving_spheres.h"
#include "box.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "C:\Users\Moritz\source\repos\stb-master/stb_image.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 color(const ray& r, hitable** world, vec3 background, int depth, hiprandState* local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	vec3 cur_emitted = vec3(0,0,0);
	for (int i = 0; i < depth; i++) {
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
				cur_attenuation *= attenuation;
				cur_emitted += emitted * cur_attenuation;
				cur_ray = scattered;
			}
			else {
				return cur_emitted + emitted * cur_attenuation;
			}
		}
		else {
			//return cur_emitted;

			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return cur_emitted; // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, hitable** world, int depth, vec3 background, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += color(r, world, background, depth, &local_rand_state);
	}
	rand_state[pixel_index] = local_rand_state;
	col /= float(ns);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);
	fb[pixel_index] = col;
}

__global__ void create_world(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		auto center2 = vec3(0, 0, -20) + vec3(0, .5, 0);
		d_list[0] = new moving_sphere(vec3(0, 0, -20), center2, 0, 1, 4, new metal(vec3(1, 0.32, 0.36), 0));
		d_list[1] = new sphere(vec3(0, -1004, -20), 1000, new lambertian(new checker_texture(vec3(0.2, 0.3, 0.1), vec3(0.9, 0.9, 0.9))));
		d_list[2] = new sphere(vec3(5, -1, -15), 2, new metal(vec3(0.90, 0.76, 0.46), 0.0));
		d_list[3] = new sphere(vec3(5, 0, -25), 3, new metal(vec3(0.65, 0.77, 0.97), 0.0));
		d_list[4] = new sphere(vec3(-5.5, 0, -15), 3, new metal(vec3(0.90, 0.90, 0.90), 0.0));

		*d_world = new hitable_list(d_list, 5);

		vec3 lookfrom(0, 0, 0);
		vec3 lookat(0, 0, -1);
		float dist_to_focus = 20;
		float aperture = .1;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			50,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);
	}
}

__global__ void create_world1(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		auto center2 = vec3(0, 0, -20) + vec3(0, .5, 0);
		auto checker = new checker_texture(vec3(0.2, 0.3, 0.1), vec3(0.9, 0.9, 0.9));
		d_list[0] = new sphere(vec3(0, -10, 0), 10, new lambertian(checker));
		d_list[1] = new sphere(vec3(0, 10, 0), 10, new lambertian(checker));

		*d_world = new hitable_list(d_list, 2);


		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			20,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);
	}
}

__global__ void create_world2(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		//hiprandState* randState;
		//hipMalloc((void**)&randState, sizeof(hiprandState));
		//
		//hiprand_init(1984, 0, 0, &randState[0]);

		auto pertext = new checker_texture(vec3(0.2, 0.3, 0.1), vec3(0.9, 0.9, 0.9));
		auto pertext1 = new noise_texture(rand_state, 4);
		d_list[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(pertext1));
		d_list[1] = new sphere(vec3(0, 2, 0), 2, new lambertian(pertext1));

		*d_world = new hitable_list(d_list, 2);


		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			20,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);

		//hipFree(randState);
	}
}

__global__ void create_world3(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		auto pertext1 = new noise_texture(rand_state, 4);
		auto difflight = new diffuse_light(vec3(4, 4, 4));
		d_list[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(pertext1));
		d_list[1] = new sphere(vec3(0, 2, 0), 2, new lambertian(pertext1));
		d_list[2] = new xy_rect(3, 5, 1, 3, -2, difflight);

		*d_world = new hitable_list(d_list, 3);


		vec3 lookfrom(26, 3, 6);
		vec3 lookat(0, 2, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			20,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);

		//hipFree(randState);
	}
}

__global__ void Earth(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, ImageTexture** tex) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		d_list[0] = new sphere(vec3(0, 0, 0), 2, new lambertian(*tex));

		*d_world = new hitable_list(d_list, 1);


		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			20,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);
	}
}

__global__ void cornell_box(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		auto red   = new lambertian   (vec3(.65, .05, .05));
		auto white = new lambertian   (vec3(.73, .73, .73));
		auto green = new lambertian   (vec3(.12, .45, .15));
		auto light = new diffuse_light(vec3(15, 15, 15));

		d_list[0] = new yz_rect(0, 555, 0, 555, 555, green);
		d_list[1] = new yz_rect(0, 555, 0, 555, 0, red);
		d_list[2] = new xz_rect(213, 343, 227, 332, 554, light);
		d_list[3] = new xz_rect(0, 555, 0, 555, 0, white);
		d_list[4] = new xz_rect(0, 555, 0, 555, 555, white);
		d_list[5] = new xy_rect(0, 555, 0, 555, 555, white);
		
		hitable* box1 = new Box(vec3(0, 0, 0), vec3(165, 330, 165), white);
		box1 = new rotate_y(box1, 15);
		box1 = new translate(box1, vec3(265, 0, 295));
		d_list[6] = box1;

		hitable* box2 = new Box(vec3(0, 0, 0), vec3(165, 165, 165), white);
		box2 = new rotate_y(box2, -18);
		box2 = new translate(box2, vec3(130, 0, 65));
		d_list[7] = box2;

		*d_world = new hitable_list(d_list, 8);


		vec3 lookfrom(278, 278, -800);
		vec3 lookat(278, 278, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			40,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);
	}
}

__global__ void cornell_box_smoke(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		auto red = new lambertian(vec3(.65, .05, .05));
		auto white = new lambertian(vec3(.73, .73, .73));
		auto green = new lambertian(vec3(.12, .45, .15));
		auto light = new diffuse_light(vec3(15, 15, 15));

		d_list[0] = new yz_rect(0, 555, 0, 555, 555, green);
		d_list[1] = new yz_rect(0, 555, 0, 555, 0, red);
		d_list[2] = new xz_rect(213, 343, 227, 332, 554, light);
		d_list[3] = new xz_rect(0, 555, 0, 555, 0, white);
		d_list[4] = new xz_rect(0, 555, 0, 555, 555, white);
		d_list[5] = new xy_rect(0, 555, 0, 555, 555, white);

		hitable* box1 = new Box(vec3(0, 0, 0), vec3(165, 330, 165), white);
		box1 = new rotate_y(box1, 15);
		box1 = new translate(box1, vec3(265, 0, 295));
		d_list[6] = new constant_medium(box1, 0.01, vec3(0, 0, 0), rand_state);

		hitable* box2 = new Box(vec3(0, 0, 0), vec3(165, 165, 165), white);
		box2 = new rotate_y(box2, -18);
		box2 = new translate(box2, vec3(130, 0, 65));
		d_list[7] = new constant_medium(box2, 0.01, vec3(1, 1, 1), rand_state);

		*d_world = new hitable_list(d_list, 8);


		vec3 lookfrom(278, 278, -800);
		vec3 lookat(278, 278, 0);
		float dist_to_focus = 10;
		float aperture = 0;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			40,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,
			1.0);
	}
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera) {
	for (int i = 0; i < 5; i++) {
		delete ((sphere*)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete* d_world;
	delete* d_camera;
}

__global__ void texture_init(unsigned char* tex_data, int nx, int ny, ImageTexture** tex) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*tex = new ImageTexture(tex_data, nx, ny);
	}
}


int main()
{
	int nx = 600;
	int ny = 600;
	int ns = 5000;
	int tx = 8;
	int ty = 8;
	int depth = 50;
	vec3 background = vec3(0.70, 0.80, 1.00);

	std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(vec3);

	int tex_x, tex_y, tex_n;
	unsigned char* tex_data_host = stbi_load("C:/Users/Moritz/source/repos/RaytracerGPU/RaytracerGPU/earthmap.jpg", &tex_x, &tex_y, &tex_n, 0);

	unsigned char* tex_data;
	checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
	checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));

	ImageTexture** texture;
	checkCudaErrors(hipMalloc((void**)&texture, sizeof(ImageTexture*)));
	texture_init << <1, 1 >> > (tex_data, tex_x, tex_y, texture);

	// allocate FB
	vec3* fb;
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	// allocate random state
	hiprandState* d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

	// make our world of hitables & the camera
	hitable** d_list;
	checkCudaErrors(hipMalloc((void**)&d_list, 5 * sizeof(hitable*)));
	hitable** d_world;
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
	camera** d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render_init << <blocks, threads >> > (nx, ny, d_rand_state);

	switch (7)
	{
	case 1:
		create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny);
		break;

	case 2:
		create_world1 << <1, 1 >> > (d_list, d_world, d_camera, nx, ny);
		break;

	case 3:
		create_world2 << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state);
		break;
	case 4:
		create_world3 << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state);
		break;
	case 5:
		cornell_box << <1, 1 >> > (d_list, d_world, d_camera, nx, ny);
		break;
	case 6:
		cornell_box_smoke << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state);
		break;
	case 7:
		Earth << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, texture);
		break;
	}

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();
	// Render our buffer
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, depth, background, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	// Output FB as Image
	std::ofstream myfile("Image1.ppm", std::ios::out | std::ios::binary);

	myfile << "P6\n" << nx << " " << ny << "\n255\n";

	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			int ir = int(255.99 * fb[pixel_index].r());
			int ig = int(255.99 * fb[pixel_index].g());
			int ib = int(255.99 * fb[pixel_index].b());

			myfile.write(reinterpret_cast<const char*>(&ir), sizeof(char));
			myfile.write(reinterpret_cast<const char*>(&ig), sizeof(char));
			myfile.write(reinterpret_cast<const char*>(&ib), sizeof(char));
		}
	}

	// clean up
	myfile.close();
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));
	free_world << <1, 1 >> > (d_list, d_world, d_camera);
	checkCudaErrors(hipGetLastError());

	hipDeviceReset();
}